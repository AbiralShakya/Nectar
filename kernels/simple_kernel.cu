#include <hip/hip_runtime.h>
#include <iostream>

__global__ void addKernel(float* a, float* b, float* c, int N) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    int N = 256;
    float *a, *b, *c;
    float *d_a, *d_b, *d_c;

    // Allocate host memory
    a = new float[N];
    b = new float[N];
    c = new float[N];

    // Initialize
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = 2 * i;
    }

    // Allocate device memory
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_c, N * sizeof(float));

    // Copy to device
    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    addKernel<<<(N + 255) / 256, 256>>>(d_a, d_b, d_c, N);

    // Copy result back
    hipMemcpy(c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    delete[] a;
    delete[] b;
    delete[] c;

    return 0;
}
